#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "minisolver.h"
#include <math.h>


extern "C" __device__ void data_output(double* vec_a,double* vec_b,int len)
{
    double error=0;
    int i;
    for(i=0;i<len;i++)
        error+=(vec_a[i]-vec_b[i])*(vec_a[i]-vec_b[i]);
    printf("gpu error:%f ",error);
}

#if __CUDA_ARCH__ < 600
__device__ double atomicDoubleAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while(assumed != old);
    return __longlong_as_double(old);
}
#endif

/*extern "C" void qrsolve(int* csr_row_ptr_a,int* csr_col_ind_a,double* csr_val_a,double* b_vec,double* x_vec,double* vec_rhs,int n,int i2)
{
    hipsolverStatus_t status;
    hipsparseStatus_t sparse_status;
    hipsolverSpHandle_t handle;
    hipsparseMatDescr_t descrA;
    int singuarity;
    int nnz=3*n-2*i2;

    status=hipsolverSpCreate(&handle);
    if(status!=HIPSOLVER_STATUS_SUCCESS)
    {
        printf("cusovler sp create error\n");
    }
    sparse_status=hipsparseCreateMatDescr(&descrA);
    if(sparse_status!=HIPSOLVER_STATUS_SUCCESS)
    {
        printf("mat descr create error\n");
    }
    status=hipsolverSpDcsrlsvqr(handle,n,nnz,descrA,csr_val_a,csr_row_ptr_a,csr_col_ind_a,b_vec,1.0e-3,0,vec_rhs,&singuarity);
    hipsolverSpDestroy(handle);
    //hipMemcpy(vec_rhs,x_vec,sizeof(double)*n,hipMemcpyDeviceToDevice);
}*/

__global__ void triang_kernel(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                            int nstride, int* stride, int* firstnode, int* lastnode, int* cellsize, int ncell)
{
    unsigned int tid;
    unsigned int thread_stride;
    int i, icellsize;
    int istride,ip;
    double p;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    thread_stride = blockDim.x * gridDim.x;
    while (tid < ncell)
    {
        icellsize = cellsize[tid];
        i = lastnode[tid];
        for(istride=nstride-1;istride>=0;--istride)
        {
            if(istride<icellsize)
            {
                ip=parent_index[i];
                p = vec_a[i]/vec_d[i];
                vec_d[ip] -= p*vec_b[i];
                vec_rhs[ip] -= p*vec_rhs[i];
                i -= stride[istride];
            }
        }
        tid += thread_stride;
    }
}

__global__ void bksub_kernel(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                            int nstride, int* stride, int* firstnode, int* lastnode, int* cellsize, int ncell)
{
    unsigned int tid, thread_stride;
    int i, icellsize;
    int istride, ip;
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    thread_stride = blockDim.x * gridDim.x;
    while (tid < ncell)
    {
        icellsize = cellsize[tid];
        i = firstnode[tid];
        vec_rhs[tid] /= vec_d[tid];
        for(istride=0;istride<icellsize;++istride)
        {
            ip=parent_index[i];
            vec_rhs[i] -= vec_b[i]*vec_rhs[ip];
            vec_rhs[i] /= vec_d[i];
            i += stride[istride+1];
        }
        //if (tid == 0)
        //    printf("%f %f\n", vec_d[0], vec_rhs[0]);
        tid += thread_stride;
    } 
}

//__global__ void solve_kernel(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
//                            int nstride, int* stride, int* firstnode, int* lastnode, int* cellsize, int ncell)
//{
//    unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
//    unsigned int thread_stride=blockDim.x*gridDim.x;
//    while(tid<ncell)
//    {
//        triang_kernel(vec_a,vec_b,vec_d,vec_rhs,parent_index,tid,cellsize[tid],nstride,stride,lastnode);
//        bksub_kernel(vec_a,vec_b,vec_d,vec_rhs,parent_index,tid,cellsize[tid],nstride,stride,firstnode);
//        tid += thread_stride;
//    }
//}

__global__ void triang(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* lastnode, int* cellsize, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip;
    double p;
    double temp;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                //printf("iorder:%d tid:%d i:%d prev_i:%d\n", iorder, tid, i, prev_node[i]);   
                ip=parent_index[i];
                //if (i < 0 || i >= 407168 || ip < 0 || ip >= 407168)
                //    printf("error icell:%d lastnode:%d i:%d ip:%d stride:%d\n", icell, lastnode[icell], i, ip, stride[istride]);
                p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                atomicAdd(vec_d + ip, temp);
                temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);
                atomicAdd(vec_rhs + ip, temp);

                i = prev_node[i];
            }
            //__syncthreads();
        }
        tid += thread_stride;
    }
}

__global__ void bksub(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                    int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                    int* firstnode, int* cellsize, int norder, int ncell, int nthread)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    ncell_per_warp = 32 / (nthread / ncell);

    while (tid < nthread)
    {
        i=firstnode[tid];
        if (tid % 32 < ncell_per_warp)
        {
            icell = tid / 32 * ncell_per_warp + tid % 32;
            //if (icell < 4)
            //    printf("tid:%d icell:%d\n", tid, icell);
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                
                vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                vec_rhs[i] /= vec_d[i];
                i = next_node[i];    
            }
            //__syncthreads();
            
        }
        tid += thread_stride;
    }
}

void __global__ triang_kernel_warp(double* vec_a,
                                double* vec_b, 
                                double* vec_d,
                                double* vec_rhs,
                                int* parent_index,
                                int* firstnode,
                                int* lastnode,
                                int* prev_node,
                                int* next_node,
                                int* max_level_per_thread,
                                int* min_level_per_thread,
                                int* children_num,
                                int threads_num,
                                int max_level,
                                int ncell,
                                int len)
{

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    int max_level_local, min_level_local;
    int i, ip, level;
    double p, temp;

    while (tid < threads_num)
    {
        max_level_local = max_level_per_thread[tid];
        min_level_local = min_level_per_thread[tid];

        i = lastnode[tid];
        for (level = max_level; level > 0; level--)
        {
            if (level <= max_level_local && level >= min_level_local)
            {
                ip = parent_index[i];
                p = vec_a[i] / vec_d[i];

                //if (ip == 826 || ip == 775)
                //    printf("\nbefore p:%f b[%d]:%f d[%d]:%f d[%d]:%f\n", p, i, vec_b[i], i, vec_d[i], ip, vec_d[ip]);

                temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                atomicAdd(vec_d + ip, temp);

                //if (ip == 826 || ip == 775)
                //    printf("after p:%f b[%d]:%f d[%d]:%f d[%d]:%f\n", p, i, vec_b[i], i, vec_d[i], ip, vec_d[ip]);

                temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);
                atomicAdd(vec_rhs + ip, temp);
                
                //__threadfence_block();
                i = prev_node[i];
            }
            __syncthreads();
        }
        tid += thread_stride;
    }
}

void __global__ bksub_kernel_warp(double* vec_a,
                                double* vec_b, 
                                double* vec_d,
                                double* vec_rhs,
                                int* parent_index,
                                int* firstnode,
                                int* lastnode,
                                int* prev_node,
                                int* next_node,
                                int* max_level_per_thread,
                                int* min_level_per_thread,
                                int* children_num,
                                int threads_num,
                                int max_level,
                                int ncell,
                                int len)
{

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    int max_level_local, min_level_local;
    int i, ip, level;
    double  temp;

    while(tid < threads_num)
    {
        max_level_local = max_level_per_thread[tid];
        min_level_local = min_level_per_thread[tid];
        i = firstnode[tid];
        if (tid == 0)
            vec_rhs[0] /= vec_d[0];
        for (level = 1; level <= max_level; level++)
        {
            if (level <= max_level_local && level >= min_level_local)
            {
                ip = parent_index[i];
                vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                vec_rhs[i] /= vec_d[i];
                i = next_node[i];
                //__threadfence_block();
            }
            __syncthreads();
        }

        tid += thread_stride;
    }
}

void __global__ solve_kernel_warp(double* vec_a,
                                double* vec_b, 
                                double* vec_d,
                                double* vec_rhs,
                                int* parent_index,
                                int* firstnode,
                                int* lastnode,
                                int* prev_node,
                                int* next_node,
                                int* max_level_per_thread,
                                int* min_level_per_thread,
                                int threads_num,
                                int max_level,
                                int ncell,
                                int len)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    int max_level_local, min_level_local;
    int i, ip, level, icell;
    double p, temp;
    //printf("tid:%d max_level:%d min_level:%d lastnode:%d firstnode:%d\n", tid, max_level_per_thread[tid], min_level_per_thread[tid], lastnode[tid], firstnode[tid]);

    while (tid < threads_num)
    {
        max_level_local = max_level_per_thread[tid];
        min_level_local = min_level_per_thread[tid];
        //printf("tid:%d firstnode:%d lastnode:%d max_level:%d, min_level:%d\n", tid, firstnode[tid], lastnode[tid], max_level_local, min_level_local);

        i = lastnode[tid];
        for (level = max_level; level > 0; level--)
        {
            //if (tid == 0)
            //{
            //    printf("level:%d i:%d prev:%d max_level:%d min_level:%d\n", level, i, prev_node[i], max_level_local, min_level_local);
            //}
            if (level <= max_level_local && level >= min_level_local)
            {
                ip = parent_index[i];
                //printf("tid:%d level:%d i:%d ip:%d firstnode:%d lastnode:%d min_level:%d max_level:%d\n", tid, level, i, ip, firstnode[tid], lastnode[tid], min_level_local, max_level_local);
                p = vec_a[i] / vec_d[i];
                //if (children_num[ip] > 1)
                //{
                    temp = -p * vec_b[i];
                    //atomicDoubleAdd(vec_d + ip, temp);
                    atomicAdd(vec_d + ip, temp);
                    temp = -p * vec_rhs[i];
                    //atomicDoubleAdd(vec_rhs + ip, temp);
                    atomicAdd(vec_rhs + ip, temp);
                //}
                //else
                //{
                //    vec_d[ip] -= p * vec_b[i];
                //    vec_rhs[ip] -= p * vec_rhs[i];
                //}
                i = prev_node[i];
            }
            __syncthreads();
        }

        i = firstnode[tid];
        if (tid % (threads_num / ncell) == 0)
        {
            icell = tid / (threads_num / ncell);
            vec_rhs[icell] /= vec_d[icell];
        }
        //if (tid == 0)
        //    vec_rhs[0] /= vec_d[0];
        for (level = 1; level <= max_level; level++)
        {
            if (level <= max_level_local && level >= min_level_local)
            {
                //if (tid == 0)
                //{
                //    printf("level:%d i:%d prev:%d\n", level, i, prev_node[i]);
                //}
                ip = parent_index[i];
                vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                vec_rhs[i] /= vec_d[i];
                i = next_node[i];
            }
            __syncthreads();
        }
        //if (tid == 0)
        //    printf("\n\n");

        tid += thread_stride;
    }
}

void solve_warp(double* vec_a,
                double* vec_b, 
                double* vec_d,
                double* vec_rhs,
                int* parent_index,
                int* firstnode,
                int* lastnode,
                int* prev_node,
                int* next_node,
                int* max_level_per_thread,
                int* min_level_per_thread,
                int threads_num,
                int max_level,
                int ncell,
                int len,
                double t,
                hipStream_t stream)
{
    unsigned int threads_per_block = threads_num / ncell;
    unsigned int block_num = (threads_num + threads_per_block - 1) / threads_per_block;

    double *a_host, *b_host, *d_host, *rhs_host;
    int *parent_host, *children_host, *firstnode_host, *lastnode_host, *min_level_host, *max_level_host, *next_host, *prev_host;
    if (t - 50 > 0 && t - 50 < 0.025)
    {
        a_host = (double*)malloc(sizeof(double) * len);
        b_host = (double*)malloc(sizeof(double) * len);
        d_host = (double*)malloc(sizeof(double) * len);
        rhs_host = (double*)malloc(sizeof(double) * len);
        parent_host = (int*)malloc(sizeof(int) * len);
        //children_host = (int*)malloc(sizeof(int) * len);
        next_host = (int*)malloc(sizeof(int) * len);
        prev_host = (int*)malloc(sizeof(int) * len);
        firstnode_host = (int*)malloc(sizeof(int) * threads_num);
        lastnode_host = (int*)malloc(sizeof(int) * threads_num);
        max_level_host = (int*)malloc(sizeof(int) * threads_num);
        min_level_host = (int*)malloc(sizeof(int) * threads_num);

        hipMemcpy(a_host, vec_a, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(b_host, vec_b, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(d_host, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(rhs_host, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(parent_host, parent_index, sizeof(int) * len, hipMemcpyDeviceToHost);
        //hipMemcpy(children_host, children_num, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(next_host, next_node, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(prev_host, prev_node, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(firstnode_host, firstnode, sizeof(int) * threads_num, hipMemcpyDeviceToHost);
        hipMemcpy(lastnode_host, lastnode, sizeof(int) * threads_num, hipMemcpyDeviceToHost);
        hipMemcpy(max_level_host, max_level_per_thread, sizeof(int) * threads_num, hipMemcpyDeviceToHost);
        hipMemcpy(min_level_host, min_level_per_thread, sizeof(int) * threads_num, hipMemcpyDeviceToHost);

        FILE* fp;
        fp = fopen("cell_data", "wb");
        fwrite(a_host, sizeof(double), len, fp);
        fwrite(b_host, sizeof(double), len, fp);
        fwrite(d_host, sizeof(double), len, fp);
        fwrite(rhs_host, sizeof(double), len, fp);
        fwrite(parent_host, sizeof(int), len, fp);
        //fwrite(children_host, sizeof(int), len, fp);
        fwrite(next_host, sizeof(int), len, fp);
        fwrite(prev_host, sizeof(int), len, fp);
        fwrite(firstnode_host, sizeof(int), threads_num, fp);
        fwrite(lastnode_host, sizeof(int), threads_num, fp);
        fwrite(max_level_host, sizeof(int), threads_num, fp);
        fwrite(min_level_host, sizeof(int), threads_num, fp);
        fclose(fp);

        fp = fopen("cell_profile", "w");
        fprintf(fp, "%d\n%d\n%d\n%d\n", ncell, len, threads_num, max_level);
        fclose(fp);
        
    }
    solve_kernel_warp<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, firstnode, lastnode, prev_node, next_node, max_level_per_thread, min_level_per_thread, threads_num, max_level, ncell, len);
    hipDeviceSynchronize();

    if (t - 50 > 0 && t - 50 < 0.025)
    {
        FILE* result_out;
        hipMemcpy(rhs_host, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(d_host, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        result_out=fopen("result_out.txt","w");
        for(int i = 0; i < len; i++)
            fprintf(result_out, "%f ", rhs_host[i]);
        fprintf(result_out, "\n");
        for(int i = 0; i < len; i++)
            fprintf(result_out, "%f ", d_host[i]);
        fclose(result_out);

    }

    //printf("block_num:%d threads_per_block:%d\n", block_num, threads_per_block);
    /*
    double rhs[2000];
    double d[2000];
    FILE* fp;
    if (t < 0.025)
    {
        hipMemcpy(rhs, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        fp = fopen("rhs_before_3.txt", "w");
        for (int i = 0; i < len; i++)
            fprintf(fp, "%f ", rhs[i]);
        fclose(fp);

        hipMemcpy(d, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        fp = fopen("d_before_3.txt", "w");
        for (int i = 0; i < len; i++)
            fprintf(fp, "%f ", d[i]);
        fclose(fp);
    }

    triang_kernel_warp<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, firstnode, lastnode, prev_node, 
                                                                next_node, max_level_per_thread, min_level_per_thread, children_num, threads_num, max_level, ncell, len);
    hipDeviceSynchronize();

    if (t < 0.025)
    {
        hipMemcpy(rhs, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        fp = fopen("rhs_after_triang_3.txt", "w");
        for (int i = 0; i < len; i++)
            fprintf(fp, "%f ", rhs[i]);
        fclose(fp);

        hipMemcpy(d, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        fp = fopen("d_after_triang_3.txt", "w");
        for (int i = 0; i < len; i++)
            fprintf(fp, "%f ", d[i]);
        fclose(fp);
    }


    bksub_kernel_warp<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, firstnode, lastnode, prev_node, 
                                                                next_node, max_level_per_thread, min_level_per_thread, children_num, threads_num, max_level, ncell, len);
    hipDeviceSynchronize();
    if (t < 0.025)
    {
        hipMemcpy(rhs, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        fp = fopen("rhs_after_3.txt", "w");
        for (int i = 0; i < len; i++)
            fprintf(fp, "%f ", rhs[i]);
        fclose(fp);

        hipMemcpy(d, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        fp = fopen("d_after_3.txt", "w");
        for (int i = 0; i < len; i++)
            fprintf(fp, "%f ", d[i]);
        fclose(fp);
    }*/

}

__global__ void kernel16_triang(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int* stride, int* map_t2c, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell, iwarp;
    double p;
    double temp;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    iwarp = tid / 32;
    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                atomicAdd(vec_d + ip, temp);
                temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);
                atomicAdd(vec_rhs + ip, temp);

                i = prev_node[i];
                //i -= stride[iwarp * (norder + 1) + iorder - 1];
            }
            //__syncthreads();
        }
        tid += thread_stride;
    }
}

__global__ void kernel16_bksub(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int* stride, int* map_t2c, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell, iwarp;
    double p;
    double temp;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    while (tid < nthread)
    {
        iwarp = tid / 32;
        icell = map_t2c[tid];
        if (icell > -1)
            vec_rhs[icell] /= vec_d[icell];
        //ncell_per_warp = 32 / (nthread / ncell);
        i = firstnode[tid];
        //if (tid % 32 < ncell_per_warp)
        //{
        //    icell = tid / 32 * ncell_per_warp + tid % 32;
        //    //if (icell < 4)
        //    //    printf("tid:%d icell:%d\n", tid, icell);
        //    if (icell < ncell)
        //        vec_rhs[icell] /= vec_d[icell];
        //}
        for(iorder = 0; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                
                //reg_rhs = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                //vec_rhs[i] = reg_rhs / vec_d[i];
                vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                vec_rhs[i] /= vec_d[i];
                i = next_node[i];    
                //i += stride[iwarp * (norder + 1) + iorder];
            }
            //__syncthreads();
            
        }
        tid += thread_stride;
    }
}

__global__ void kernel16(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int* stride, int* map_t2c, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell, iwarp;
    double p;
    double temp;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    iwarp = tid / 32;
    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                atomicAdd(vec_d + ip, temp);
                temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);
                atomicAdd(vec_rhs + ip, temp);

                //i = prev_node[i];
                i -= stride[iwarp * (norder + 1) + iorder - 1];
            }
            //__syncthreads();
        }

        icell = map_t2c[tid];
        if (icell > -1)
            vec_rhs[icell] /= vec_d[icell];
        //ncell_per_warp = 32 / (nthread / ncell);
        i = firstnode[tid];
        //if (tid % 32 < ncell_per_warp)
        //{
        //    icell = tid / 32 * ncell_per_warp + tid % 32;
        //    //if (icell < 4)
        //    //    printf("tid:%d icell:%d\n", tid, icell);
        //    if (icell < ncell)
        //        vec_rhs[icell] /= vec_d[icell];
        //}
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                
                //reg_rhs = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                //vec_rhs[i] = reg_rhs / vec_d[i];
                vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                vec_rhs[i] /= vec_d[i];
                //i = next_node[i];    
                i += stride[iwarp * (norder + 1) + iorder];
            }
            //__syncthreads();
            
        }
        //if (tid == 0)
        //    printf("%f %f\n", vec_d[0], vec_rhs[0]);
        tid += thread_stride;
    }
}

__global__ void kernel16_tp(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int* stride, int* map_t2c, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell, iwarp;
    double p;
    double temp;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    iwarp = tid / 32;
    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                atomicAdd(vec_d + ip, temp);
                temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);
                atomicAdd(vec_rhs + ip, temp);

                i = prev_node[i];
                //i -= stride[iwarp * (norder + 1) + iorder - 1];
            }
            //__syncthreads();
        }

        //icell = map_t2c[tid];
        //if (icell > -1)
        //    vec_rhs[icell] /= vec_d[icell];
        ncell_per_warp = 32 / (nthread / ncell);
        i = firstnode[tid];
        if (tid % 32 < ncell_per_warp)
        {
            icell = tid / 32 * ncell_per_warp + tid % 32;
            //if (icell < 4)
            //    printf("tid:%d icell:%d\n", tid, icell);
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                
                //reg_rhs = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                //vec_rhs[i] = reg_rhs / vec_d[i];
                vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                vec_rhs[i] /= vec_d[i];
                i = next_node[i];    
                //i += stride[iwarp * (norder + 1) + iorder];
            }
            //__syncthreads();
            
        }
        //if (tid == 0)
        //    printf("%f %f\n", vec_d[0], vec_rhs[0]);
        tid += thread_stride;
    }
}


void solve_branch(double* vec_a, 
                  double* vec_b, 
                  double* vec_d, 
                  double* vec_rhs, 
                  int* parent_index,
                  int* prev_node,
                  int* next_node,
                  int* max_order_per_thread,
                  int* min_order_per_thread,
                  int* firstnode,
                  int* lastnode,
                  int* stride,
                  int* map_t2c,
                  int ncell,
                  int nthread,
                  int norder,
                  int len,
                  double t,
                  hipStream_t stream)
{
    unsigned int threads_per_block = 128;
    unsigned int block_num = (nthread + threads_per_block - 1) / threads_per_block;
    double *a_host = NULL,*b_host = NULL, *d_host = NULL, *rhs_host = NULL;
    int *parent_host = NULL, *firstnode_host = NULL, *lastnode_host = NULL, *prev_node_host = NULL, *next_node_host = NULL, *branchsize_host = NULL;
    int *stride_host = NULL;
    int *max_order_host = NULL, *min_order_host = NULL;
    bool *reg_avai_host = NULL;
    int nwarps = 0;
    
    /*if (t - 50 > 0 && t - 50 < 0.025)
    {
        a_host = (double*)malloc(sizeof(double) * len);
        b_host = (double*)malloc(sizeof(double) * len);
        d_host = (double*)malloc(sizeof(double) * len);
        rhs_host = (double*)malloc(sizeof(double) * len);
        //branchsize_host = (int*)malloc(sizeof(int) * nthread);
        parent_host = (int*)malloc(sizeof(int) * len);
        prev_node_host = (int*)malloc(sizeof(int) * len);
        next_node_host = (int*)malloc(sizeof(int) * len);
        max_order_host = (int*)malloc(sizeof(int) * nthread);
        min_order_host = (int*)malloc(sizeof(int) * nthread);
        firstnode_host = (int*)malloc(sizeof(int) * nthread);
        lastnode_host = (int*)malloc(sizeof(int) * nthread);
        //reg_avai_host = (bool*)malloc(sizeof(bool) * len);

        hipMemcpy(a_host, vec_a, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(b_host, vec_b, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(d_host, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(rhs_host, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        //hipMemcpy(branchsize_host, branchsize, sizeof(int) * nthread, hipMemcpyDeviceToHost);
        hipMemcpy(parent_host, parent_index, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(prev_node_host, prev_node, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(next_node_host, next_node, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(max_order_host, max_order_per_thread, sizeof(int) * nthread, hipMemcpyDeviceToHost);
        hipMemcpy(min_order_host, min_order_per_thread, sizeof(int) * nthread, hipMemcpyDeviceToHost);
        hipMemcpy(firstnode_host, firstnode, sizeof(int) * nthread, hipMemcpyDeviceToHost);
        hipMemcpy(lastnode_host, lastnode, sizeof(int) * nthread, hipMemcpyDeviceToHost);
        //hipMemcpy(reg_avai_host, reg_avai, sizeof(bool) * nthread * (norder + 1), hipMemcpyDeviceToHost);

        FILE *fp;
        fp = fopen("cell_data", "wb");
        fwrite(a_host, sizeof(double), len, fp);
        fwrite(b_host, sizeof(double), len, fp);
        fwrite(d_host, sizeof(double), len, fp);
        fwrite(rhs_host, sizeof(double), len, fp);
        fwrite(parent_host, sizeof(int), len, fp);
        fwrite(prev_node_host, sizeof(int), len, fp);
        fwrite(next_node_host, sizeof(int), len, fp);
        fwrite(max_order_host, sizeof(int), nthread, fp);
        fwrite(min_order_host, sizeof(int), nthread, fp);
        fwrite(firstnode_host, sizeof(int), nthread, fp);
        fwrite(lastnode_host, sizeof(int), nthread, fp);
        //fwrite(branchsize_host, sizeof(int), nthread, fp);
        fclose(fp);

        //fp = fopen("reg_avai", "wb");
        //fwrite(reg_avai_host, sizeof(bool), nthread * (norder + 1), fp);
        //fclose(fp);

        fp = fopen("cell_profile", "w");
        fprintf(fp, "%d\n%d\n%d\n%d\n", ncell, len, nthread, norder);
        fclose(fp);

        if (nthread % 32 == 0)
            nwarps = nthread / 32;
        else 
            nwarps = nthread / 32 + 1;
        stride_host = (int*)malloc(sizeof(int) * nwarps * (norder + 1));
        hipMemcpy(stride_host, stride, sizeof(int) * nwarps * (norder + 1), hipMemcpyDeviceToHost);
        fp = fopen("stride_array", "wb");
        fwrite(stride_host, sizeof(int), nwarps * (norder + 1), fp);
        fclose(fp);
    }*/
    
    //hipDeviceSynchronize();
    kernel16<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, prev_node, next_node, max_order_per_thread, min_order_per_thread, firstnode, lastnode, stride, map_t2c, norder, ncell, nthread, t);
    //kernel16_tp<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, prev_node, next_node, max_order_per_thread, min_order_per_thread, firstnode, lastnode, stride, map_t2c, norder, ncell, nthread, t);
    
    hipDeviceSynchronize();
    //FILE *fp = NULL;
    //fp = fopen("solve3", "ab");
    //d_host = (double*)malloc(sizeof(double) * len);
    //rhs_host = (double*)malloc(sizeof(double) * len);
    //hipMemcpy(d_host, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
    //hipMemcpy(rhs_host, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
    //fwrite(d_host, sizeof(double), len, fp);
    //fwrite(rhs_host, sizeof(double), len, fp);
    //fclose(fp); 

    
    /*if (t - 50 > 0 && t - 50 < 0.025)
    {
        FILE* result_out;
        hipMemcpy(rhs_host, vec_rhs, sizeof(double) * len, hipMemcpyDeviceToHost);
        hipMemcpy(d_host, vec_d, sizeof(double) * len, hipMemcpyDeviceToHost);
        result_out=fopen("result_out.txt","w");
        for(int i = 0; i < len; i++)
            fprintf(result_out, "%f ", rhs_host[i]);
        fprintf(result_out, "\n");
        for(int i = 0; i < len; i++)
            fprintf(result_out, "%f ", d_host[i]);
    
        if (result_out)
            fclose(result_out);
        if (a_host)
            free(a_host);
        if (b_host)
            free(b_host);
        if (d_host)
            free(d_host);
        if (rhs_host)
            free(rhs_host);
        if (parent_host)
            free(parent_host);
        if (prev_node_host)
            free(prev_node_host);
        if (next_node_host)
            free(next_node_host);
        if (firstnode_host)
            free(firstnode_host);
        if (lastnode_host)
            free(lastnode_host);
        if (branchsize_host)
            free(branchsize_host);

    }*/
}


void cusolve_interleaved1(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int nstride, int* stride, int* firstnode, int* lastnode, int* cellsize, int ncell,
                        int len,double t, hipStream_t stream)
{
    unsigned int threads_per_block=128;
    unsigned int block_num=(ncell+threads_per_block-1)/threads_per_block;

    double* a_host,*b_host,*d_host,*rhs_host;
    int* parent_host, *stride_host,*firstnode_host,*lastnode_host,*cellsize_host;
    FILE* fp,*result_out;
    int i;
    
    if(t-50>0 && t-50<0.025)
    {
        a_host=(double*)malloc(sizeof(double)*len);    
        b_host=(double*)malloc(sizeof(double)*len);
        d_host=(double*)malloc(sizeof(double)*len);
        rhs_host=(double*)malloc(sizeof(double)*len);
        parent_host=(int*)malloc(sizeof(int)*len);
        stride_host=(int*)malloc(sizeof(int)*(nstride+1));
        firstnode_host=(int*)malloc(sizeof(int)*ncell);
        lastnode_host=(int*)malloc(sizeof(int)*ncell);
        cellsize_host=(int*)malloc(sizeof(int)*ncell);

        hipMemcpy(a_host,vec_a,sizeof(double)*len,hipMemcpyDeviceToHost);
        hipMemcpy(b_host,vec_b,sizeof(double)*len,hipMemcpyDeviceToHost);
        hipMemcpy(d_host,vec_d,sizeof(double)*len,hipMemcpyDeviceToHost);
        hipMemcpy(rhs_host,vec_rhs,sizeof(double)*len,hipMemcpyDeviceToHost);
        hipMemcpy(parent_host,parent_index,sizeof(int)*len,hipMemcpyDeviceToHost);
        hipMemcpy(stride_host,stride,sizeof(int)*(nstride+1),hipMemcpyDeviceToHost);
        hipMemcpy(firstnode_host,firstnode,sizeof(int)*ncell,hipMemcpyDeviceToHost);
        hipMemcpy(lastnode_host,lastnode,sizeof(int)*ncell,hipMemcpyDeviceToHost);
        hipMemcpy(cellsize_host,cellsize,sizeof(int)*ncell,hipMemcpyDeviceToHost);
    
        fp=fopen("cell_data","wb");
        fwrite(a_host,sizeof(double),len,fp);
        fwrite(b_host,sizeof(double),len,fp);
        fwrite(d_host,sizeof(double),len,fp);
        fwrite(rhs_host,sizeof(double),len,fp);
        fwrite(parent_host,sizeof(int),len,fp);
        fwrite(stride_host,sizeof(int),nstride+1,fp);
        fwrite(firstnode_host,sizeof(int),ncell,fp);
        fwrite(lastnode_host,sizeof(int),ncell,fp);
        fwrite(cellsize_host,sizeof(int),ncell,fp);
        
        printf("*****%d\n\n\n*****",len);
        printf("*****%d\n\n\n*****",ncell);
        printf("*****%d\n\n\n*****",nstride);
        fclose(fp);

        fp = fopen("cell_profile", "w");
        fprintf(fp, "%d\n%d\n%d\n", ncell, len, nstride);
        fclose(fp);
    }

    hipDeviceSynchronize();
    triang_kernel<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, nstride, stride, firstnode, lastnode, cellsize, ncell);
    bksub_kernel<<<block_num, threads_per_block, 0, stream>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, nstride, stride, firstnode, lastnode, cellsize, ncell);
    hipDeviceSynchronize();

    if(t-50>0&&t-50<0.025)
    {
        hipMemcpy(rhs_host,vec_rhs,sizeof(double)*len,hipMemcpyDeviceToHost);
        hipMemcpy(d_host,vec_d,sizeof(double)*len,hipMemcpyDeviceToHost);
        result_out=fopen("result_out.txt","w");
        for(i=0;i<len;i++)
            fprintf(result_out,"%f ",rhs_host[i]);
        fprintf(result_out,"\n");
        for(i=0;i<len;i++)
            fprintf(result_out,"%f ",d_host[i]);
    
        fclose(result_out);
        free(a_host);
        free(b_host);
        free(d_host);
        free(rhs_host);
        free(parent_host);
        free(stride_host);
        free(firstnode_host);
        free(lastnode_host);
        free(cellsize_host);

    }
}













